#include <hip/hip_runtime.h>
#pragma warning(disable : 4324)
#include <cub/device/device_select.cuh>
#pragma warning(default : 4324)
#include "cub_select.h"

template <typename T>
void cubDeviceSelectUnique(std::span<const T> inKeys, std::span<T> outKeys, uint32_t* pNumOutKeys, hipStream_t stream)
{
    size_t tmpStorageSize = 0;
    void* pTmpStorage = nullptr;
    hipcub::DeviceSelect::Unique(pTmpStorage, tmpStorageSize, inKeys.data(), outKeys.data(), pNumOutKeys, outKeys.size(), stream);
    hipMallocAsync(&pTmpStorage, tmpStorageSize, stream);
    hipcub::DeviceSelect::Unique(pTmpStorage, tmpStorageSize, inKeys.data(), outKeys.data(), pNumOutKeys, outKeys.size(), stream);
    hipFreeAsync(pTmpStorage, stream);
}

template void cubDeviceSelectUnique(std::span<const uint32_t>, std::span<uint32_t>, uint32_t*, hipStream_t);
