#include <hip/hip_runtime.h>
#pragma warning(disable : 4324)
// #include <cub/device/device_scan.cuh>
#include <cub/device/device_merge_sort.cuh>
#include <cub/device/device_radix_sort.cuh>
#include <thrust/iterator/transform_output_iterator.h>
#pragma warning(default : 4324)
//
#include "cub_merge_sort.h"
#include "cuda_error_check.h"
#include "cuda_helpers_cpp.h"

#include "dags/my_gpu_dags/my_gpu_hash_dag/hash_tables/individual_chaining_hash_table.h"

template <typename T1, typename T2>
void cubDeviceMergeSortPairs(std::span<T1> keys, std::span<T2> items, hipStream_t stream)
{
    size_t requiredMemorySize = 0;
    hipcub::DeviceMergeSort::SortPairs(nullptr, requiredMemorySize, thrust::raw_pointer_cast(keys.data()), thrust::raw_pointer_cast(items.data()), keys.size(), thrust::less<T1>(), stream);
    void* pMemory = nullptr;
    hipMallocAsync(&pMemory, requiredMemorySize, stream);
    hipcub::DeviceMergeSort::SortPairs(pMemory, requiredMemorySize, thrust::raw_pointer_cast(keys.data()), thrust::raw_pointer_cast(items.data()), keys.size(), thrust::less<T1>(), stream);
    hipFreeAsync(pMemory, stream);
}

//template void cubDeviceMergeSortPairs(std::span<typename IntermediateSVO::Node>, std::span<uint32_t>, hipStream_t);
//template void cubDeviceMergeSortPairs(std::span<typename IntermediateSVO::Leaf>, std::span<uint32_t>, hipStream_t);

/* template void cubDeviceMergeSortPairs(std::span<MyGpuDagElement<2>>, std::span<uint32_t>, hipStream_t);
template void cubDeviceMergeSortPairs(std::span<MyGpuDagElement<3>>, std::span<uint32_t>, hipStream_t);
template void cubDeviceMergeSortPairs(std::span<MyGpuDagElement<4>>, std::span<uint32_t>, hipStream_t);
template void cubDeviceMergeSortPairs(std::span<MyGpuDagElement<5>>, std::span<uint32_t>, hipStream_t);
template void cubDeviceMergeSortPairs(std::span<MyGpuDagElement<6>>, std::span<uint32_t>, hipStream_t);
template void cubDeviceMergeSortPairs(std::span<MyGpuDagElement<7>>, std::span<uint32_t>, hipStream_t);
template void cubDeviceMergeSortPairs(std::span<MyGpuDagElement<8>>, std::span<uint32_t>, hipStream_t);
template void cubDeviceMergeSortPairs(std::span<MyGpuDagElement<9>>, std::span<uint32_t>, hipStream_t);
template void cubDeviceMergeSortPairs(std::span<MyGpuDagElement<10>>, std::span<uint32_t>, hipStream_t);*/
