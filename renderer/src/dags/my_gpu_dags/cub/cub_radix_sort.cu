#include <hip/hip_runtime.h>
#pragma warning(disable : 4324)
#include <cub/device/device_radix_sort.cuh>
#pragma warning(default : 4324)
#include "cub_radix_sort.h"

template <typename T>
void cubDeviceRadixSortKeys(std::span<const T> inKeys, std::span<T> outKeys, hipStream_t stream)
{
    size_t tmpStorageSize = 0;
    void* pTmpStorage = nullptr;
    hipcub::DeviceRadixSort::SortKeys(pTmpStorage, tmpStorageSize, inKeys.data(), outKeys.data(), inKeys.size(), 0, sizeof(T) * 8, stream);
    hipMallocAsync(&pTmpStorage, tmpStorageSize, stream);
    hipcub::DeviceRadixSort::SortKeys(pTmpStorage, tmpStorageSize, inKeys.data(), outKeys.data(), inKeys.size(), 0, sizeof(T) * 8, stream);
    hipFreeAsync(pTmpStorage, stream);
}

template void cubDeviceRadixSortKeys(std::span<const uint32_t>, std::span<uint32_t>, hipStream_t);
